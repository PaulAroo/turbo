// Copyright 2021 Pierre Talbot, Frédéric Pinel

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <new>
#include <chrono>

#include "solver.cuh"
#include "vstore.cuh"
#include "constraints.cuh"
#include "cuda_helper.hpp"
#include "statistics.cuh"
#include "status.cuh"
#include "search.cuh"

template<typename T>
T* cons_alloc(std::vector<T> &c)
{
  T* props;
  CUDIE(hipMallocManaged(&props, c.size() * sizeof(T)));
  for (int i=0; i < c.size(); ++i) {
    new(props + i) T(c[i]);
  }
  return props;
}

template <typename T>
CUDA_DEVICE
bool propagate(T* constraints, int nc, VStore& vstore, PropagatorsStatus& pstatus) {
  bool has_changed = false;
  for(int i=0; i<nc; ++i) {
    T& p = constraints[i];
    bool has_changed2 = p.propagate(vstore);
    has_changed |= has_changed2;
    Status s = has_changed2 ? UNKNOWN : IDLE;
    if(p.is_entailed(vstore)) {
      s = ENTAILED;
    }
    if(p.is_disentailed(vstore)) {
      s = DISENTAILED;
    }
    pstatus.inplace_join(p.uid, s);
  }
  return has_changed;
}

CUDA_GLOBAL void propagate_nodes_k(
    TreeData* td,
    TemporalProp* tem_p, int nt,
    ReifiedLogicalAnd* rei_p, int nr,
    LinearIneq* lin_p, int nl) {
  int nid = threadIdx.x + blockIdx.x * blockDim.x;
  bool has_changed = true;
  PropagatorsStatus& pstatus = *(td->node_array[nid].pstatus);
  VStore& vstore = *(td->node_array[nid].vstore);
  while(has_changed && pstatus.join() < ENTAILED) {
    has_changed = propagate(tem_p, nt, vstore, pstatus);
    has_changed |= propagate(rei_p, nr, vstore, pstatus);
    has_changed |= propagate(lin_p, nl, vstore, pstatus);
  }
  // We propagate once more to verify that all propagators are really entailed.
  if(pstatus.join() == ENTAILED) {
    propagate(tem_p, nt, vstore, pstatus);
    propagate(rei_p, nr, vstore, pstatus);
    propagate(lin_p, nl, vstore, pstatus);
  }
  /*
  propagate_k<TemporalProp><<<constraints.temporal.size(), 1>>>(shared_data, tem_p);
  propagate_k<LinearIneq><<<constraints.linearIneq.size(), 1>>>(shared_data, lin_p);
  propagate_k<ReifiedLogicalAnd><<<constraints.reifiedLogicalAnd.size(), 1>>>(shared_data, rei_p);
  CUDIE(hipDeviceSynchronize());
  */
}

CUDA_GLOBAL void transfer_search(TreeData* td) {
    td->transferFromSearch();
}

void solve(VStore* vstore, Constraints constraints, Var minimize_x, int timeout)
{
  INFO(constraints.print(*vstore));

  Var* temporal_vars = constraints.temporal_vars(vstore->size());

  TreeData *tree_data;
  CUDIE(hipMallocManaged(&tree_data, sizeof(*tree_data)));
  new(tree_data) TreeData(temporal_vars, minimize_x, *vstore, constraints.size());

  auto tem_p = cons_alloc<TemporalProp>(constraints.temporal);
  auto rei_p = cons_alloc<ReifiedLogicalAnd>(constraints.reifiedLogicalAnd);
  auto lin_p = cons_alloc<LinearIneq>(constraints.linearIneq);
  auto t1 = std::chrono::high_resolution_clock::now();

  while (!tree_data->stack.is_empty()) {
    auto current = std::chrono::high_resolution_clock::now();
    if (std::chrono::duration_cast<std::chrono::seconds>(current - t1).count() > timeout) {
      break;
    }
    tree_data->transferFromSearch();

    propagate_nodes_k<<<tree_data->node_array.size(), 1>>>(
        tree_data,
        tem_p, constraints.temporal.size(),
        rei_p, constraints.reifiedLogicalAnd.size(),
        lin_p ,constraints.linearIneq.size());
    CUDIE(hipDeviceSynchronize());
    tree_data->transferToSearch();
  }

  auto t2 = std::chrono::high_resolution_clock::now();
  CUDIE(hipFree(tem_p));
  CUDIE(hipFree(rei_p));
  CUDIE(hipFree(lin_p));
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();

  tree_data->stats.print();
  if(duration > timeout * 1000) {
    std::cout << "solveTime=timeout" << std::endl;
  }
  else {
    std::cout << "solveTime=" << duration << std::endl;
  }

  CUDIE(hipFree(tree_data));
  CUDIE(hipFree(temporal_vars));
}
