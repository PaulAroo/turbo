// Copyright 2021 Pierre Talbot, Frédéric Pinel

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <new>

#include "solver.cuh"
#include "vstore.cuh"
#include "constraints.cuh"
#include "cuda_helper.hpp"

CUDA_VAR bool Exploring = true;

const int PROPS_TYPE = 3;
const int PROP_OPS = 3;
const int MAX_DEPTH_TREE = 100;

struct PropagatorsStatus {
  bool* entailed;
  bool* disentailed;
  bool* idle;
  int n;
  PropagatorsStatus(int n) {
    this->n = n;
    CUDIE(hipMallocManaged(&entailed, n*sizeof(bool)));
    CUDIE(hipMallocManaged(&disentailed, n*sizeof(bool)));
    CUDIE(hipMallocManaged(&idle, n*sizeof(bool)));
    for(int i = 0; i < n; ++i) {
      entailed[i] = false;
      disentailed[i] = false;
      idle[i] = false;
    }
  }

  __host__ PropagatorsStatus(const PropagatorsStatus& other) {
    n = other.n;
    CUDIE(hipMallocManaged(&entailed, n*sizeof(bool)));
    CUDIE(hipMallocManaged(&disentailed, n*sizeof(bool)));
    CUDIE(hipMallocManaged(&idle, n*sizeof(bool)));
    // switch to memcpy on device
    for(int i = 0; i < n; ++i) {
      entailed[i] = other.entailed[i];
      disentailed[i] = other.disentailed[i];
      // NOTE: no need to copy "idle", must actually be initialized at false for the next node.
    }
  }

  void free() {
    CUDIE(hipFree(entailed));
    CUDIE(hipFree(disentailed));
    CUDIE(hipFree(idle));
  }

  CUDA bool all(bool* array) {
    for(int i = 0; i < n; ++i) {
      if(!array[i]) {
        return false;
      }
    }
    return true;
  }


  CUDA bool any(bool* array) {
    for(int i = 0; i < n; ++i) {
      if(array[i]) {
        printf("Propagator %d disentailed.\n", i);
        return true;
      }
    }
    return false;
  }

  CUDA bool all_entailed() {
    return all(entailed);
  }

  CUDA bool any_disentailed() {
    return any(disentailed);
  }

  CUDA bool all_idle() {
    return all(idle);
  }

  CUDA void wake_up_all() {
    for(int i = 0; i < n; ++i) {
      idle[i] = false;
    }
  }
};

template <typename T>
struct Engine {
  PropagatorsStatus* status;
  VStore* vstore;
  T* props;

  CUDA Engine(PropagatorsStatus* status, VStore* vstore, T* props)
    : status(status), vstore(vstore), props(props) {}

  CUDA inline void updateIsEntailed(size_t id) {
    T& p = props[id];
    status->entailed[p.uid] = p.is_entailed(*vstore);
  }

  CUDA inline void updateIsDisentailed(size_t id) {
    T& p = props[id];
    status->disentailed[p.uid] = p.is_disentailed(*vstore);
  }

  CUDA inline void propagate(size_t id) {
    T& p = props[id];
    status->idle[p.uid] = !p.propagate(*vstore);
    //printf("Propagate %lu\n", id);
  }
};

// Select the variable with the smallest domain in the store.
CUDA Var first_fail(VStore& vstore, Var* vars) {
  Var x = -1;
  int lowest_lb = limit_max();
  for(int k = 0; vars[k] != -1; ++k) {
    int i = vars[k];
    if (vstore[i].lb < lowest_lb && !vstore[i].is_assigned()) {
      x = i;
      lowest_lb = vstore[i].lb;
    }
  }
  assert(x != -1);
  return x;
}

CUDA void assign_lb(VStore& vstore, Var x) {
  vstore.update(x, {vstore[x].lb, vstore[x].lb});
}

CUDA Interval not_assign_lb(VStore& vstore, Var x) {
  return {vstore[x].lb + 1, vstore[x].ub};
}

struct BacktrackingFrame {
  VStore vstore;
  Var var;
  Interval itv;
};

CUDA_GLOBAL void search(PropagatorsStatus* status, VStore* current, VStore* best_sol, Var minimize_x, Var* temporal_vars) {
  printf("starting search\n");
  BacktrackingFrame* stack = new BacktrackingFrame[MAX_DEPTH_TREE];
  size_t stack_size = 0;
  Interval best_bound = {limit_min(), limit_max()};
  while (Exploring) {
    bool all_entailed = status->all_entailed();
    bool any_disentailed = status->any_disentailed();
    if ((any_disentailed || all_entailed) != current->all_assigned()) {
      printf("invariant inconsistent.\n");
    }
    if (status->all_idle() && !(all_entailed || any_disentailed)) {
      printf("All IDLE, depth = %d\n", stack_size);
      BacktrackingFrame frame;
      frame.var = first_fail(*current, temporal_vars);
      frame.itv = not_assign_lb(*current, frame.var);
      frame.vstore = *current;
      stack[stack_size] = std::move(frame);
      ++stack_size;
      assert(stack_size < MAX_DEPTH_TREE);
      printf("Branching: %d = %d..%d ",
        frame.var, (*current)[frame.var].lb, (*current)[frame.var].ub);
      assign_lb(*current, frame.var);
      printf(" -> %d..%d \\/ %d..%d\n", (*current)[frame.var].lb, (*current)[frame.var].ub,
        frame.itv.lb, frame.itv.ub);
      status->wake_up_all();
    }
    else if(all_entailed || any_disentailed) {
      if(any_disentailed) {
        printf("backtracking on failed node...\n");
      }
      else if(all_entailed) {
        best_bound = (*current)[minimize_x];
        best_bound.ub = best_bound.lb;
        printf("backtracking on solution...(bound = %d)\n", best_bound.ub);
        best_bound.lb = limit_min();
        *best_sol = *current;
      }
      // If nothing is left in the stack, we stop the search, it means we explored the full search tree.
      if(stack_size == 0) {
        Exploring = false;
      }
      else {
        BacktrackingFrame& frame = stack[stack_size - 1];
        --stack_size;
        // Commit to the branch.
        frame.vstore.update(frame.var, frame.itv);
        // Adjust the objective.
        frame.vstore.update(minimize_x, frame.vstore[minimize_x].join(best_bound));
        // Swap the current branch with the backtracked one.
        *current = frame.vstore;
        // Change the IDLE status of propagators.
        status->wake_up_all();
      }
    }
  }
  delete[] stack;
  printf("stop search\n");
}

template<typename T>
CUDA_GLOBAL void entail_k(Engine<T>* engine) {
  size_t id = threadIdx.x + blockIdx.x*blockDim.x;
  while (Exploring) {
    engine->updateIsEntailed(id);
  }
}

template<typename T>
CUDA_GLOBAL void disentail_k(Engine<T>* engine) {
  size_t id = threadIdx.x + blockIdx.x*blockDim.x;
  while (Exploring) {
    engine->updateIsDisentailed(id);
  }
}

template<typename T>
CUDA_GLOBAL void propagate_k(Engine<T>* engine) {
  size_t id = threadIdx.x + blockIdx.x*blockDim.x;
  while (Exploring) {
    engine->propagate(id);
  }
}

// The status and vstore are shared among all propagators of all types.
// The UID inside a propagator, e.g., `TemporalProp::uid`, refers to the index of the propagator in the various arrays of `status`.
template<typename T>
Engine<T>* launch(PropagatorsStatus* status, VStore* vstore, std::vector<T> &c, hipStream_t s[PROP_OPS])
{
  // printf("launching %lu threads on stream %p\n", c.size(), s[0]);

  T* props;
  CUDIE(hipMallocManaged(&props, c.size() * sizeof(T)));
  for (int i=0; i < c.size(); ++i) {
    props[i] = c[i];
  }

  Engine<T> *engine;
  CUDIE(hipMallocManaged(&engine, sizeof(Engine<T>)));
  *engine = Engine<T>(status, vstore, props);

  propagate_k<T><<<1, c.size(), 0, s[0]>>>(engine);
  entail_k<T><<<1, c.size(), 0, s[1]>>>(engine);
  disentail_k<T><<<1, c.size(), 0, s[2]>>>(engine);
  CUDIE0();
  return engine;
}

void solve(VStore* vstore, Constraints constraints, Var minimize_x, const char** var2name_raw)
{
  // std::cout << "Before propagation: " << std::endl;
  // vstore->print(var2name_raw);

  void* status_raw;
  CUDIE(hipMallocManaged(&status_raw, sizeof(PropagatorsStatus)));
  PropagatorsStatus* status = new(status_raw) PropagatorsStatus(constraints.size());

  void* best_sol_raw;
  CUDIE(hipMallocManaged(&best_sol_raw, sizeof(VStore)));
  VStore* best_sol = new(best_sol_raw) VStore();

  Var* temporal_vars = constraints.temporal_vars(vstore->size());

  hipStream_t monitor;
  CUDIE(hipStreamCreate(&monitor));
  hipStream_t streams[PROPS_TYPE][PROP_OPS];
  for (int i=0; i < PROPS_TYPE; ++i) {
    for (int j=0; j < PROP_OPS; ++j) {
      CUDIE(hipStreamCreate(&streams[i][j]));
    }
  }

  search<<<1,1,0,monitor>>>(status, vstore, best_sol, minimize_x, temporal_vars);
  CUDIE0();

  auto engines_0 = launch<TemporalProp>(status, vstore, constraints.temporal, streams[0]);
  CUDIE0();
  auto engines_1 = launch<ReifiedLogicalAnd>(status, vstore, constraints.reifiedLogicalAnd, streams[1]);
  CUDIE0();
  auto engines_2 = launch<LinearIneq>(status, vstore, constraints.linearIneq, streams[2]);
  CUDIE0();

  CUDIE(hipDeviceSynchronize());

  if(best_sol->size() == 0) {
    printf("Could not find a solution.\n");
  }
  else {
    printf("Best bound found is %d.\n", (*best_sol)[minimize_x].lb);
    // best_sol->print(var2name_raw);
    best_sol->free();
  }
  CUDIE(hipFree(best_sol_raw));
  CUDIE(hipFree(temporal_vars));

  status->free();
  CUDIE(hipFree(status_raw));

  CUDIE(hipFree(engines_0->props));
  CUDIE(hipFree(engines_0));
  CUDIE(hipFree(engines_1->props));
  CUDIE(hipFree(engines_1));
  CUDIE(hipFree(engines_2->props));
  CUDIE(hipFree(engines_2));

  CUDIE(hipStreamDestroy(monitor));
  for (int i=0; i < PROPS_TYPE; ++i) {
    for (int j=0; j < PROP_OPS; ++j) {
      CUDIE(hipStreamDestroy(streams[i][j]));
    }
  }
}
