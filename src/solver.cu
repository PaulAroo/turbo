#include "hip/hip_runtime.h"
// Copyright 2021 Pierre Talbot, Frédéric Pinel

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <new>

#include "solver.cuh"
#include "vstore.cuh"
#include "constraints.cuh"
#include "cuda_helper.hpp"

__global__
void is_active_k() {
  __shared__ uint curr;
  while (1) {
    asm("nanosleep.u32 10000000;");
    curr = Act_cnt;
    if (curr <= 0) {
      printf("no activity\n");
      break;
    } else {
      printf("active! (%u)\n", curr);
    }
  }
}

template<typename Constraint>
CUDA_GLOBAL void propagate_k(Constraint *c, VStore* vstore) {
  int ic = threadIdx.x + blockIdx.x*blockDim.x;
  bool worked, preworked = 0;
  while (Exploring) {
    worked = c[ic].propagate(*vstore);
    if (!preworked && worked) { ++Act_cnt; }
    else if (preworked && !worked) { --Act_cnt; }
    preworked = worked;
  }
}

template<typename ConstraintT>
ConstraintT* launch(std::vector<ConstraintT> &c, hipStream_t s, VStore *vstore) {
  printf("launching %d threads on stream %d\n", c.size(), s);
  ConstraintT *constraints;
  CUDIE(hipMallocManaged(&constraints, c.size()*sizeof(ConstraintT)));
  for (int i=0; i<c.size(); ++i) {
    constraints[i] = c[i];
  }
  propagate_k<ConstraintT><<<1, c.size(), 0, s>>>(constraints, vstore);
  return constraints;
}

void solve(VStore* vstore, Constraints constraints, const char** var2name_raw) {
  vstore->print(var2name_raw);

  hipStream_t monitor;
  CUDIE(hipStreamCreate(&monitor));
  const int NCT = 3;
  hipStream_t sConstraint[NCT];
  for (int i=0; i<NCT; ++i) {
    CUDIE(hipStreamCreate(&sConstraint[i]));
  }
  auto c0 = launch<XplusYleqC>(constraints.xPlusYleqC, sConstraint[0], vstore);
  auto c1 = launch<ReifiedLogicalAnd>(constraints.reifiedLogicalAnd, sConstraint[1], vstore);
  auto c2 = launch<LinearIneq>(constraints.linearIneq, sConstraint[2], vstore);
  
  is_active_k<<<1,1,0,monitor>>>();
  CUDIE0();

  CUDIE(hipDeviceSynchronize());

  printf("\n\nAfter propagation:\n");
  vstore->print(var2name_raw);

  CUDIE(hipFree(c0));
  CUDIE(hipFree(c1));
  CUDIE(hipFree(c2));
  CUDIE(hipStreamDestroy(monitor));
  for (int i=0; i<NCT; ++i) {
    CUDIE(hipStreamDestroy(sConstraint[i]));
  }
}
