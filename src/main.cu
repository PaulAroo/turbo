#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <cmath.h>

#define CUDIE(result) { \
        hipError_t e = (result); \
        if (e != hipSuccess) { \
                std::cerr << __FILE__ << ":" << __LINE__; \
                std::cerr << " CUDA runtime error: " << hipGetErrorString(e) << '\n'; \
                exit((int)e); \
        }}

#define CUDIE0() CUDIE(hipGetLastError())

typedef size_t Var;

struct Interval {
  int lb;
  int ub;

  inline void join(Interval b) {
    lb = max(lb, b.lb);
    ub = min(ub, b.ub);
  }

  bool operator==(int x) {
    return lb == x && ub == x;
  }
};

struct VStore {
  Interval* data;
  size_t size;

  VStore(int nvar) {
    size = nvar;
    CUDIE(hipMallocManaged(&data, sizeof(Interval) * nvar));
  }

  void print_store() {
    for(int i=0; i < size; ++i) {
      printf("%d = [%d..%d]\n", i, data[i].lb, data[i].ub);
    }
  }

  // lb <= x <= ub
  void dom(Var x, Interval itv) {
    data[x] = itv;
  }

  Interval& operator[](const size_t i) {
    return data[i];
  }
};

/// x + y <= c
struct XplusYleqC {
  Var x;
  Var y;
  int c;

  XplusYleqC(Var x, Var y, int c) : x(x), y(y), c(c) {}

  void propagate(VStore vstore)
  {
    vstore[x].join({vstore[x].lb, c - vstore[y].lb});
    vstore[y].join({vstore[y].lb, c - vstore[x].lb});
  }

  bool is_entailed(VStore vstore) {
    return vstore[x].ub + vstore[y].ub <= c;
  }

  bool is_disentailed(VStore vstore) {
    return vstore[x].lb + vstore[y].lb > c;
  }
};

// /// b <=> left /\ right
// struct ReifiedLogicalAnd {
//   Var b;
//   XplusYleqC left;
//   XplusYleqC right;

//   ReifiedLogicalAnd(Var b, XplusYleqC left, XplusYleqC right) :
//     b(b), left(left), right(right) {}

//   void propagate(VStore vstore) {
//     if vstore[b] == 0 {

//     }
//     else if vstore[b] == 1 {
//       left.propagate(vstore);
//       right.propagate(vstore);
//     }
//     else if left.is_entailed(vstore) && right.is_entailed(vstore) {
//       vstore[b] = 1;
//     }
//     else if left.is_disentailed(vstore) && right.is_disentailed(vstore) {
//       vstore[b] = 0;
//     }
//   }
// }

int main() {
  VStore vstore = new_vstore(2);
  int x = 0;
  int y = 1;
  dom(vstore, x, {0, 2});
  dom(vstore, y, {1, 3});
  print_store(vstore);
  x_plus_y_leq_c(vstore, x, y, 2);
  print_store(vstore);
  return 0;
}
