#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <cmath.h>

#define CUDIE(result) { \
        hipError_t e = (result); \
        if (e != hipSuccess) { \
                std::cerr << __FILE__ << ":" << __LINE__; \
                std::cerr << " CUDA runtime error: " << hipGetErrorString(e) << '\n'; \
                exit((int)e); \
        }}

#define CUDIE0() CUDIE(hipGetLastError())

template<typename T>__device__ __host__ T min(T a, T b) { return a<=b ? a : b; }
template<typename T>__device__ __host__ T max(T a, T b) { return a>=b ? a : b; }
typedef size_t Var;

struct Interval {
  int lb;
  int ub;

  __host__ __device__
  void join(Interval b) {
    lb = max<int>(lb, b.lb);
    ub = min<int>(ub, b.ub);
  }

  __host__ __device__
  bool operator==(int x) {
    return lb == x && ub == x;
  }
};

struct VStore {
  Interval* data;
  size_t size;

  VStore(int nvar) {
    size = nvar;
    CUDIE(hipMallocManaged(&data, sizeof(*data) * nvar));
  }

  VStore(const VStore& s) {
    // use : size{s.size}, ... ?
    size = s.size;
    data = s.data;
  }

  void print_store() {
    for(int i=0; i < size; ++i) {
      printf("%d = [%d..%d]\n", i, data[i].lb, data[i].ub);
    }
  }

  // lb <= x <= ub
  void dom(Var x, Interval itv) {
    data[x] = itv;
  }

  Interval& operator[](const size_t i) {
    return data[i];
  }
};

/// x + y <= c
struct XplusYleqC {
  Var x;
  Var y;
  int c;

  XplusYleqC(Var x, Var y, int c) : x(x), y(y), c(c) {}

  __device__ __host__
  void propagate(VStore vstore)
  {
    vstore[x].join({vstore[x].lb, c - vstore[y].lb});
    vstore[y].join({vstore[y].lb, c - vstore[x].lb});
  }

  __device__ __host__
  bool is_entailed(VStore vstore) {
    return vstore[x].ub + vstore[y].ub <= c;
  }

  __device__ __host__
  bool is_disentailed(VStore vstore) {
    return vstore[x].lb + vstore[y].lb > c;
  }
};

__global__ void propagate_k(struct XplusYleqC xpylc, VStore vstore) {
	xpylc.propagate(vstore);
}



// /// b <=> left /\ right
// struct ReifiedLogicalAnd {
//   Var b;
//   XplusYleqC left;
//   XplusYleqC right;

//   ReifiedLogicalAnd(Var b, XplusYleqC left, XplusYleqC right) :
//     b(b), left(left), right(right) {}

//   void propagate(VStore vstore) {
//     if vstore[b] == 0 {

//     }
//     else if vstore[b] == 1 {
//       left.propagate(vstore);
//       right.propagate(vstore);
//     }
//     else if left.is_entailed(vstore) && right.is_entailed(vstore) {
//       vstore[b] = 1;
//     }
//     else if left.is_disentailed(vstore) && right.is_disentailed(vstore) {
//       vstore[b] = 0;
//     }
//   }
// }

int main() {
  VStore vstore(2);
  int x = 0;
  int y = 1;
  dom(*vstore, x, {0, 2});
  dom(*vstore, y, {1, 3});
  print_store(*vstore);
  x_plus_y_leq_c<<<1,1>>>(vstore, x, y, 2);
  CUDIE0();
  CUDIE(hipDeviceSynchronize());
  // page fault expected:
  print_store(*vstore);
  return 0;
}
